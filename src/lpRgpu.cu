#include "hip/hip_runtime.h"
#include "lpRgpu.cuh"
#include "main_kernels.cuh"
#include "simple_kernels.cuh"


//init global parameters
lpRgpu::lpRgpu(float* params, int Nparams)
{  
	readGlobalParametersArr(params);
	err = hipMalloc((void **)&derho, Ntheta*Nrho*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
	err = hipMalloc((void **)&dfl, Nslices*Ntheta*Nrho*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
	err = hipMalloc((void **)&dflc, Nslices*Ntheta_R2C*Nrho*sizeof(float2)); if (err!=0) callErr(hipGetErrorString(err));

	//init rho space
	err = hipMemcpy(derho,erho,Ntheta*Nrho*sizeof(float),hipMemcpyHostToDevice); if (err!=0) callErr(hipGetErrorString(err));

	hipChannelFormatDesc texf_desc = hipCreateChannelDesc<float>();	
	hipExtent volumeSize = make_hipExtent(Ntheta,Nrho,Nslices); 
	
	err = hipMalloc3DArray(&dfla, &texf_desc,volumeSize,hipArrayLayered); if (err!=0) callErr(hipGetErrorString(err));
	texfl.addressMode[0] = hipAddressModeWrap;
	texfl.addressMode[1] = hipAddressModeWrap;
	texfl.filterMode = hipFilterModeLinear;
	texfl.normalized  = true;
	hipBindTextureToArray(texfl, dfla,texf_desc);  

	//fft plans for Nslices slices
	hipfftResult res1,res2;
	int ffts[] = {Nrho,Ntheta};
	int idist = Nrho*Ntheta;int odist = Nrho*(Ntheta/2+1);
	int inembed[] = {Nrho, Ntheta};int onembed[] = {Nrho, Ntheta/2+1};
	res1 = hipfftPlanMany(&plan_forward, 2, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_R2C, Nslices); if (res1!=0) {char errs[16];sprintf(errs,"fwd hipfftPlanMany error %d",res1);callErr(errs);}
	res2 = hipfftPlanMany(&plan_inverse, 2, ffts, onembed, 1, odist, inembed, 1, idist, HIPFFT_C2R, Nslices); if (res2!=0) {char errs[16];sprintf(errs,"inv hipfftPlanMany error %d",res1);callErr(errs);}

	err = hipMalloc((void **)&dR, Nslices*Nproj*N*sizeof(float));	if (err!=0) callErr(hipGetErrorString(err));
	err = hipMalloc((void **)&df, Nslices*N*N*sizeof(float));	if (err!=0) callErr(hipGetErrorString(err));
	dfZfwd = NULL;//for checking if fwd is created
}
lpRgpu::~lpRgpu()
{
	delete[] erho;
	//free gpu memory
	hipFree(derho);
	hipFree(dfl);
	hipFree(dflc);
	hipUnbindTexture(texfl);
	hipFreeArray(dfla);	
	hipfftDestroy(plan_forward);
	hipfftDestroy(plan_inverse);
	hipFree(df);
	hipFree(dR);

	//delete parameters for fwd and adj transform if they are initialized
	if(dfZfwd) deleteFwd();
	deleteAdj();
}

//init parameters for forward (Radon) tranform 
void lpRgpu::initFwd(int* paramsi, int Nparamsi, float* paramsf, int Nparamsf)
{
	fgs = new fwdgrids(Nspan);
	readFwdParametersArr(paramsi,paramsf);

	fgs->initgpu();
	err = hipMalloc((void **)&dfZfwd, Ntheta_R2C*Nrho*sizeof(float2)); if (err!=0) callErr(hipGetErrorString(err));
	err = hipMalloc((void **)&dtmpf, Nslices*N*N*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));//delete to do

	//copy Fourier transform of Z	
	err = hipMemcpy(dfZfwd,fZfwd,Ntheta_R2C*Nrho*sizeof(float2),hipMemcpyHostToDevice); if (err!=0) callErr(hipGetErrorString(err));

	hipChannelFormatDesc texf_desc = hipCreateChannelDesc<float>();
	hipExtent volumeSize = make_hipExtent(N,N,Nslices); 
	err = hipMalloc3DArray(&dfa, &texf_desc, volumeSize,hipArrayLayered); if (err!=0) callErr(hipGetErrorString(err));
	texf.addressMode[0] = hipAddressModeWrap;
	texf.addressMode[1] = hipAddressModeWrap;	
	texf.filterMode = hipFilterModeLinear;
	texf.normalized = true;
	hipBindTextureToArray(texf, dfa,texf_desc);

	//init result with zeros
	err = hipMemset(dR, 0, Nslices*Nproj*N*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
}
void lpRgpu::deleteFwd()
{
	delete[] fZfwd;
	hipFree(dtmpf);
	hipUnbindTexture(texf);
	hipFreeArray(dfa);
	hipFree(dfZfwd);
	delete fgs;
	err = hipGetLastError(); if(err!=0) callErr(hipGetErrorString(err));
}

//init parameters for adjoint tranform (back-projection)
void lpRgpu::initAdj(int* paramsi, int Nparamsi, float* paramsf, int Nparamsf)
{
	ags = new adjgrids(Nspan);
	readAdjParametersArr(paramsi,paramsf);
	ags->initgpu();

	err = hipMalloc((void **)&dfZadj, Ntheta_R2C*Nrho*sizeof(float2)); if (err!=0) callErr(hipGetErrorString(err));
	err = hipMalloc((void **)&dtmpR, Nslices*N*Nproj*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));//delete to do

	//copy Fourier transform of adj Z
	hipMemcpy(dfZadj,fZadj,Ntheta_R2C*Nrho*sizeof(float2),hipMemcpyHostToDevice);

	hipChannelFormatDesc texf_desc = hipCreateChannelDesc<float>();	
	hipExtent volumeSize = make_hipExtent(Nproj,N,Nslices); 
	err = hipMalloc3DArray(&dRa, &texf_desc, volumeSize,hipArrayLayered); if (err!=0) callErr(hipGetErrorString(err));
	texR.addressMode[0] = hipAddressModeWrap;
	texR.addressMode[1] = hipAddressModeWrap;
	texR.filterMode = hipFilterModeLinear;
	texR.normalized = true;
	hipBindTextureToArray(texR, dRa,texf_desc);

	//init result with zeros
	err = hipMemset(df, 0, Nslices*N*N*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));

	//init filter
	if (filter)
	{
		int osfilter = 4;
		err = hipMalloc((void **)&dfilter, N*osfilter*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
		err = hipMemcpy(dfilter, filter,N*osfilter*sizeof(float),hipMemcpyDefault); if (err!=0) callErr(hipGetErrorString(err));
	
		hipfftPlan1d(&plan_f_forward,N*osfilter,HIPFFT_C2C,Nproj);
		hipfftPlan1d(&plan_f_inverse,N*osfilter,HIPFFT_C2C,Nproj);
		err = hipMalloc((void **)&dRt, Nproj*N*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
		err = hipMalloc((void **)&dRc, Nproj*N*osfilter*sizeof(float2)); if (err!=0) callErr(hipGetErrorString(err));
	}
}
void lpRgpu::deleteAdj()
{
	delete[] fZadj;
	hipFree(dtmpR);
	hipFree(dfZadj);
	hipUnbindTexture(texR);
	hipFreeArray(dRa);
	delete ags;

	if(filter)
	{
		delete[] filter;
		hipFree(dfilter);
		hipFree(dRt);
		hipFree(dRc);
		hipfftDestroy(plan_f_forward);
		hipfftDestroy(plan_f_inverse);
	}	
}

hipError_t copy3DDeviceToArray(hipArray* dfa, float* df, hipExtent ext)
{
	hipMemcpy3DParms param = { 0 };
	param.srcPtr   = make_hipPitchedPtr((void*)df, ext.width*sizeof(float), ext.width, ext.height);
	param.dstArray = dfa;
	param.kind = hipMemcpyDeviceToDevice;
	param.extent = ext;
	return hipMemcpy3D(&param);
}

hipError_t copy3Dshifted(float *dst, int dstx,int dsty, hipExtent dstext, float* src, int srcx, int srcy, hipExtent srcext, hipExtent copyext)
{
	hipMemcpy3DParms param = { 0 };
	param.srcPtr = make_hipPitchedPtr(&src[srcy*srcext.width+srcx], srcext.width*sizeof(float), srcext.width, srcext.height);
	param.dstPtr = make_hipPitchedPtr(&dst[dsty*dstext.width+dstx], dstext.width*sizeof(float), dstext.width, dstext.height);
	param.kind = hipMemcpyDefault;
	copyext.width*= sizeof(float);
	param.extent = copyext;
	return hipMemcpy3D(&param);
}


//compute Radon transform for several slices
void lpRgpu::execFwdMany(float* R, int Nslices2_, int N_, int Nproj_, float* f, int Nslices1_, int N2_, int N1_)
{
	hipMemset(df,0,N*N*Nslices*sizeof(float));
	err = copy3Dshifted(df,N/2-N1_/2,N/2-N2_/2,make_hipExtent(N,N,Nslices),f,0,0,make_hipExtent(N1_, N2_, Nslices1_),make_hipExtent(N1_,N2_,Nslices1_)); if(err!=0) callErr(hipGetErrorString(err));  	    execFwd();
	int shift = N_/2-cor;
    err = copy3Dshifted(R,0,0,make_hipExtent(Nproj_,N_,Nslices2_),dR,0,N/2-N_/2+shift,make_hipExtent(Nproj, N, Nslices),make_hipExtent(Nproj_,N_,Nslices2_)); if(err!=0) callErr(hipGetErrorString(err));
}

//compute back-projection for several slices
void lpRgpu::execAdjMany(float* f, int Nslices1_, int N2_, int N1_, float* R, int Nslices2_, int N_, int Nproj_)
{
	hipMemset(dR,0,Nproj*N*Nslices*sizeof(float));
	int shift = N_/2-cor;
        err = copy3Dshifted(dR,0,N/2-N_/2+shift,make_hipExtent(Nproj, N, Nslices),R,0,0,make_hipExtent(Nproj_,N_,Nslices2_),make_hipExtent(Nproj_,N_,Nslices2_)); if(err!=0) callErr(hipGetErrorString(err));   	   
	padding(N_,shift);
	applyFilter();
	execAdj();
        err = copy3Dshifted(f,0,0,make_hipExtent(N1_, N2_, Nslices1_),df,N/2-N1_/2,N/2-N2_/2,make_hipExtent(N,N,Nslices),make_hipExtent(N1_,N2_,Nslices1_)); if(err!=0) callErr(hipGetErrorString(err));  }

//padding
void lpRgpu::padding(int N_, int shift)
{
	uint GS31 = (uint)ceil(Nproj/(float)MBS21);uint GS32 = (uint)ceil(N/(float)MBS22);uint GS33 = (uint)ceil(Nslices/(float)MBS33);
       dim3 dimBlock(MBS31,MBS32,MBS33);dim3 dimGrid(GS31,GS32,GS33);
        padker<<<dimGrid,dimBlock>>>(dR,N/2-N_/2+shift,N/2+N_/2+shift-1,Nproj,N,Nslices);
}

//prefilter to compensate amplitudes in cubic interpolation
void lpRgpu::prefilter2D(float *df, float* dtmpf, uint width, uint height)
{
	//transpose for optimal cache usage
	uint GS31 = (uint)ceil(width/(float)MBS31);uint GS32 = (uint)ceil(height/(float)MBS32);uint GS33 = (uint)ceil(Nslices/(float)MBS33);
	dim3 dimBlock(MBS31,MBS32,MBS33);dim3 dimGrid(GS31,GS32,GS33);
	transpose<<<dimGrid,dimBlock>>>(dtmpf, df,width, height,Nslices);

	//compensate in samples for x direction
	uint GS41 = (uint)ceil(height/(float)MBS41);uint GS42 = (uint)ceil(Nslices/(float)MBS42); 
	dim3 dimBlock1(MBS41,MBS42);dim3 dimGrid1(GS41,GS42);
	SamplesToCoefficients2DY<<<dimGrid1, dimBlock1>>>(dtmpf,height*sizeof(float),height, width,Nslices);

	//transpose back
	GS31 = (uint)ceil(height/(float)MBS31);GS32 = (uint)ceil(width/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);
	dim3 dimBlock2(MBS31,MBS32,MBS33);dim3 dimGrid2(GS31,GS32,GS33);
	transpose<<<dimGrid2,dimBlock2>>>(df,dtmpf,height, width,Nslices);

	//compensate in samples for y direction
	GS41 = (uint)ceil(width/(float)MBS41);GS42 = (uint)ceil(Nslices/(float)MBS42); 
	dim3 dimBlock3(MBS41,MBS42);dim3 dimGrid3(GS41,GS42);	
	SamplesToCoefficients2DY<<<dimGrid3, dimBlock3>>>(df,width*sizeof(float),width,height,Nslices);
}

//compute Radon transform in log-polar coordinates
void lpRgpu::execFwd()
{
	err = hipMemset(dtmpf, 0, Nslices*N*N*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
	err = hipMemset(dR, 0, Nslices*Nproj*N*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
	//compensation for cubic interpolation
	if(interp_type) prefilter2D(df,dtmpf,N,N);

	//init gpu array with binded texture
	copy3DDeviceToArray(dfa,df,make_hipExtent(N, N, Nslices));

	//CUDA block and grid sizes
	dim3 dimBlock(MBS31,MBS32,MBS33);
	uint GS31, GS32, GS33;

	for(int k = 0;k<Nspan;k++)
	{
		err = hipMemset(dfl, 0, Nslices*Ntheta*Nrho*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err)); 

		//interp Cartesian to log-polar grid
		GS31 = (uint)ceil(ceil(sqrtf((float)fgs->Ncidsfwd))/(float)MBS31);GS32 = (uint)ceil(ceil(sqrtf((float)fgs->Ncidsfwd))/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid(GS31,GS32,GS33);
		interp<<<dimGrid, dimBlock>>>(0+interp_type*3,dfl,fgs->dlp2C1[k],fgs->dlp2C2[k],MBS31*GS31,fgs->Ncidsfwd,N,N,Nslices,fgs->dcidsfwd,Ntheta*Nrho);
		
		//multiplication e^{\rho}
		GS31 = (uint)ceil(Ntheta/(float)MBS31);GS32 = (uint)ceil(Nrho/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid1(GS31,GS32,GS33);
		mulexp<<<dimGrid1, dimBlock>>>(dfl,derho,Ntheta,Nrho, Nslices);

		//forward FFT
		hipfftExecR2C(plan_forward, (hipfftReal*)dfl,(hipfftComplex*)dflc);

		//multiplication by fZ
		GS31 = (uint)ceil(Ntheta_R2C/(float)MBS31);GS32 = (uint)ceil(Nrho/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid2(GS31,GS32,GS33);
		mul<<<dimGrid2, dimBlock>>>(1/(float)(Ntheta*Nrho),dflc,dfZfwd,Ntheta_R2C,Nrho,Nslices);

		//inverse FFT
		hipfftExecC2R(plan_inverse,(hipfftComplex*)dflc,(hipfftReal*)dfl);

		//init gpu array with binded texture
		copy3DDeviceToArray(dfla,dfl,make_hipExtent(Ntheta, Nrho, Nslices));

		//interp log-polar to polar grid
		GS31 = (uint)ceil(ceil(sqrtf((float)fgs->Npids[k]))/(float)MBS31);GS32 = (uint)ceil(ceil(sqrtf((float)fgs->Npids[k]))/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid3(GS31,GS32,GS33);
		interp<<<dimGrid3, dimBlock>>>(2+interp_type*3,dR,fgs->dp2lp1[k],fgs->dp2lp2[k],MBS31*GS31,fgs->Npids[k],Ntheta,Nrho,Nslices,fgs->dpids[k],Nproj*N);
	}
}

//compute back-projection in log-polar coordinates
void lpRgpu::execAdj()
{
	hipMemset(dtmpR, 0, Nslices*Nproj*N*sizeof(float)); 
	hipMemset(df, 0, Nslices*N*N*sizeof(float)); 
	//compensation for cubic interpolation
	prefilter2D(dR,dtmpR,Nproj,N);
	//init gpu array with binded texture
	copy3DDeviceToArray(dRa,dR,make_hipExtent(Nproj, N, Nslices));

	//CUDA block and grid sizes
	dim3 dimBlock(MBS31,MBS32,MBS33);
	uint GS31, GS32, GS33;
	for(int k = 0;k<Nspan;k++)
	{   
		hipMemset(dfl, 0, Nslices*Ntheta*Nrho*sizeof(float)); 
		//interp from polar to log-polar grid
		GS31 = (uint)ceil(ceil(sqrt(ags->Nlpidsadj))/(float)MBS31); GS32 = (uint)ceil(ceil(sqrt(ags->Nlpidsadj))/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid(GS31,GS32,GS33);
		interp<<<dimGrid, dimBlock>>>(1+interp_type*3,dfl,ags->dlp2p1[k],ags->dlp2p2[k],MBS31*GS31,ags->Nlpidsadj,Nproj,N,Nslices,ags->dlpidsadj,Ntheta*Nrho);

		//interp from polar to log-polar grid additional points
		GS31 = (uint)ceil(ceil(sqrt(ags->Nwids))/(float)MBS31); GS32 = (uint)ceil(ceil(sqrt(ags->Nwids))/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid4(GS31,GS32,GS33);
		interp<<<dimGrid4, dimBlock>>>(1+interp_type*3,dfl,ags->dlp2p1w[k],ags->dlp2p2w[k],MBS31*GS31,ags->Nwids,Nproj,N,Nslices,ags->dwids,Ntheta*Nrho);

		//Forward FFT
		hipfftExecR2C(plan_forward, (hipfftReal*)dfl,(hipfftComplex*)dflc);

		//multiplication by adjoint fZ
		GS31 = (uint)ceil(Ntheta_R2C/(float)MBS31); GS32 = (uint)ceil(Nrho/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid2(GS31,GS32,GS33);
		mul<<<dimGrid2, dimBlock>>>(1/(float)(Ntheta*Nrho),dflc,dfZadj,Ntheta_R2C,Nrho,Nslices);

		//Inverse FFT
		hipfftExecC2R(plan_inverse,(hipfftComplex*)dflc,(hipfftReal*)dfl);

		//init gpu array with binded texture
		copy3DDeviceToArray(dfla,dfl,make_hipExtent(Ntheta, Nrho, Nslices));

		//interp from log-polar to Cartesian grid
		GS31 = (uint)ceil(ceil(sqrt(ags->Ncidsadj))/(float)MBS31); GS32 = (uint)ceil(ceil(sqrt(ags->Ncidsadj))/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid3(GS31,GS32,GS33);
		interp<<<dimGrid3, dimBlock>>>(2+interp_type*3,df,ags->dC2lp1[k],ags->dC2lp2[k],MBS31*GS31,ags->Ncidsadj,Ntheta,Nrho,Nslices,ags->dcidsadj,N*N);
	}
}

//apply filter in frequency
void lpRgpu::applyFilter()
{
	if (!filter) return;
	
	dim3 dimBlock(MBS21,MBS22);
	uint GS21, GS22;
	int osfilter = 4;
	for(int ij = 0;ij<Nslices;ij++)
	{
		hipMemset(dRc, 0, 2*Nproj*N*osfilter*sizeof(float));
		//transpose data
		GS21 = ceil(Nproj/(float)MBS21);GS22 = ceil(N/(float)MBS22);dim3 dimGrid(GS21,GS22);
		transpose<<<dimGrid, dimBlock>>>(dRt, &dR[Nproj*N*ij], Nproj, N,1);
		
		//copy to complex array
		GS21 = ceil(N/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid1(GS21,GS22);	
		copyc<<<dimGrid1, dimBlock>>>(dRt,dRc,N,Nproj,osfilter);

		//fftshift 
		GS21 = ceil(N*osfilter/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid2(GS21,GS22);
		fftshift<<<dimGrid2, dimBlock>>>(dRc,N*osfilter,Nproj);

		//forward fft	
		hipfftExecC2C(plan_f_forward,dRc,dRc,HIPFFT_FORWARD);
	
		//fftshift
		GS21 = ceil(N*osfilter/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid3(GS21,GS22);
		fftshift<<<dimGrid3, dimBlock>>>(dRc,N*osfilter,Nproj);

		//mulfilter
		GS21 = ceil(N*osfilter/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid4(GS21,GS22);
		mulfilter<<<dimGrid4, dimBlock>>>(dRc,dfilter,N*osfilter,Nproj);

		//fftshift
		GS21 = ceil(N*osfilter/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid5(GS21,GS22);
		fftshift<<<dimGrid5, dimBlock>>>(dRc,N*osfilter,Nproj);
	
		//adjoint fft	
		hipfftExecC2C(plan_f_inverse,dRc,dRc,HIPFFT_BACKWARD);
	
		//fftshift
		GS21 = ceil(N*osfilter/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid6(GS21,GS22);
		fftshift<<<dimGrid6, dimBlock>>>(dRc,N*osfilter,Nproj);

		//copy from complex array
		GS21 = ceil(N*osfilter/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid7(GS21,GS22);	
		copycback<<<dimGrid7, dimBlock>>>(dRt,dRc,N,Nproj,osfilter);

		//tranpose back
		GS21 = ceil(N/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid8(GS21,GS22);
		transpose<<<dimGrid8, dimBlock>>>(&dR[Nproj*N*ij], dRt, N,Nproj,1);

		//mul const
		GS21 = ceil(Nproj/(float)MBS21);GS22 = ceil(N/(float)MBS22);dim3 dimGrid9(GS21,GS22);
		mulconst<<<dimGrid9, dimBlock>>>(&dR[Nproj*N*ij],1/(float)(osfilter*N), Nproj, N);
	}
}

